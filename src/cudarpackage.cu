#include "header/cuda_usage.h"
#include "header/cholesky.h"
#include "header/quad_form.h"
#include "header/summary2.h"
#include "header/chain.h"
#include "header/iterator.h"
#include "header/construct_prec.h"
#include "header/distribution.h"
#include "header/cluster_probability.h"
#include "header/printing.h"
#include "header/gibbs.h"
#include "header/wrap_R.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <R.h>
#include <Rinternals.h>
#include <Rmath.h>
// This prevents the replacement of "beta" by Rmath.h
#ifdef beta
#undef beta
#endif
#include <boost/progress.hpp>

extern "C" SEXP RgetDeviceCount(){
  int count = 0;
  hipGetDeviceCount(&count);
  
  SEXP result = PROTECT(allocVector(INTSXP, 1));
  INTEGER(result)[0] = count;
  UNPROTECT(1);
  
  return result;
}

extern "C" SEXP RsetDevice(SEXP device) {
  SEXP result = PROTECT(allocVector(INTSXP, 1));
  INTEGER(result)[0] = setDevice(INTEGER(device)[0]);
  UNPROTECT(1);
  return result;
}

extern "C" SEXP RgetDevice(){
  int device = 0;
  hipGetDevice(&device);
  
  SEXP result = PROTECT(allocVector(INTSXP, 1));
  INTEGER(result)[0] = device;
  UNPROTECT(1);
  
  return result;
}

//wrapper to chol_multiple which does in-place cholesky decomposition on a (flattened) array of matrices
extern "C" SEXP Rchol_multiple(SEXP all, SEXP arraydim, SEXP n_array){
  int dim = INTEGER(arraydim)[0];
  int reps = INTEGER(n_array)[0];
  double *aptr = REAL(all);
  fvec_d dvec(aptr, aptr + length(all));
  realIter begin = dvec.begin();
  realIter end = dvec.end();
  chol_multiple(begin, end, dim, reps);
  fvec_h hvec(dvec.begin(), dvec.end());
  SEXP out = PROTECT(allocVector(REALSXP, length(all)));
  for(int i=0; i<length(all); ++i)
    REAL(out)[i] = hvec[i];
  UNPROTECT(1);
  return out;
}

extern "C" SEXP Rconstruct_prec(SEXP Rdata, SEXP Rpriors, SEXP Rchain){
  data_t data = Rdata_wrap(Rdata);
  priors_t priors = Rpriors_wrap(Rpriors);
  chain_t chain = Rchain_wrap(Rchain);
  
  int psize = priors.K * data.V * data.V;
  summary2 summary(priors.K, chain.zeta, data);
  fvec_d prec(psize);
  construct_prec(prec, data, priors, chain, summary.Mk, 0);

  SEXP out_prec = PROTECT(allocVector(REALSXP, psize));
  for(int i=0; i<psize; ++i)
    REAL(out_prec)[i] = prec[i];
  UNPROTECT(1);
  return out_prec;
}

extern "C" SEXP Rgamma_rng(SEXP Rseed, SEXP a, SEXP b, bool logscale=false){

  int n = length(a), seed = INTEGER(Rseed)[0];
  
  //instantiate RNGs
  hiprandState *devStates;
  CUDA_CALL(hipMalloc((void **) &devStates, n * sizeof(hiprandState)));
  
  //temporary memory
  
  fvec_d out_d(n);
  fvec_h out_h(n);
 
  fvec_d a_d(REAL(a), REAL(a)+n);
  fvec_d b_d(REAL(b), REAL(b)+n);
  
  double *out_d_ptr = thrust::raw_pointer_cast(out_d.data());
  double *a_d_ptr = thrust::raw_pointer_cast(a_d.data());
  double *b_d_ptr = thrust::raw_pointer_cast(b_d.data());
    
  //set up RNGs
  int n_blocks = n/512 + 1;
  setup_kernel<<<n_blocks,512>>>(seed, n, devStates);
  
  //sample from Gamma(a, b)
  getGamma<<<n_blocks,512>>>(devStates, n, a_d_ptr, b_d_ptr, out_d_ptr, logscale);
  
  //copy to host
  thrust::copy(out_d.begin(), out_d.end(), out_h.begin());
  
  //transfer memory
  SEXP out = PROTECT(allocVector(REALSXP, n));
  for(int i=0; i<n; ++i)
    REAL(out)[i] = out_h[i];
  
  //clean up
  CUDA_CALL(hipFree(devStates));
  UNPROTECT(1);
  
  return out;
}

extern "C" SEXP Rbeta_rng(SEXP Rseed, SEXP a, SEXP b){

  int n = length(a), seed = INTEGER(Rseed)[0];

  //instantiate RNGs
  hiprandState *devStates;
  CUDA_CALL(hipMalloc((void **) &devStates, n * sizeof(hiprandState)));
  
  //temporary memory
  
  fvec_d out_d(n);
  fvec_h out_h(n);
 
  fvec_d a_d(REAL(a), REAL(a)+n);
  fvec_d b_d(REAL(b), REAL(b)+n);
  
  double *out_d_ptr = thrust::raw_pointer_cast(out_d.data());
  double *a_d_ptr = thrust::raw_pointer_cast(a_d.data());
  double *b_d_ptr = thrust::raw_pointer_cast(b_d.data());
    
  //set up RNGs
  int blocksize = 512;
  int n_blocks = n/blocksize + 1;
  setup_kernel<<<n_blocks,blocksize>>>(seed, n, devStates);
  
  //sample from Beta(a, b)
  getBeta<<<n_blocks,blocksize>>>(devStates, n, a_d_ptr, b_d_ptr, out_d_ptr, false);
  
  //copy to host
  thrust::copy(out_d.begin(), out_d.end(), out_h.begin());
  
  //transfer memory
  SEXP out = PROTECT(allocVector(REALSXP, n));
  for(int i=0; i<n; ++i)
    REAL(out)[i] = out_h[i];
  
  //clean up
  CUDA_CALL(hipFree(devStates));
  UNPROTECT(1);
  
  return out;
}

extern "C" SEXP Rquad_form_multi(SEXP A, SEXP x, SEXP n, SEXP dim){

  double *Aptr = REAL(A), *xptr = REAL(x);
  int N = INTEGER(n)[0], D = INTEGER(dim)[0];

  fvec_d dA(Aptr, Aptr+D*D);
  fvec_d dx(xptr, xptr+N*D);
  fvec_d dy(N);

  quad_form_multi(dA, dx, dy, N, D);

  SEXP y = PROTECT(allocVector(REALSXP, N));
  for(int i=0; i<N; ++i)
    REAL(y)[i] = dy[i];

  UNPROTECT(1);
  return y;
}

extern"C" SEXP Rsummary2(SEXP zeta, SEXP ytyR, SEXP ytxR, SEXP xtyR, SEXP G, SEXP V, SEXP K){
  int g = INTEGER(G)[0], v = INTEGER(V)[0], k = INTEGER(K)[0];
  int *zp = INTEGER(zeta);
  fvec_h xtx(v*v, 1.0);
  double *ytyp = REAL(ytyR);
  double *ytxp = REAL(ytxR);
  double *xtyp = REAL(xtyR);
  double *xtxp = &(xtx[0]);
  data_t data(ytyp, xtyp, xtxp, g, v, 1);
  
  ivec_d ZETA(zp, zp+g);
  summary2 smry(k, ZETA, data);
  
  /*smry.print_Mk();
  smry.print_yty();
  smry.print_xty();*/
  
  SEXP out = PROTECT(allocVector(VECSXP, 4));
  SEXP OCCo = PROTECT(allocVector(INTSXP, 1));
  SEXP ytyo = PROTECT(allocVector(REALSXP, smry.num_occupied));
  SEXP ytxo = PROTECT(allocVector(REALSXP, smry.num_occupied*v));
  SEXP xtyo = PROTECT(allocVector(REALSXP, smry.num_occupied*v));
  INTEGER(OCCo)[0] = smry.num_occupied;
  
  for(int i=0; i<smry.num_occupied; ++i){
    REAL(ytyo)[i] = smry.yty_sums[i];
  }
  
  for(int i=0; i<smry.num_occupied*v; ++i){
    REAL(ytxo)[i] = smry.ytx_sums[i];
    REAL(xtyo)[i] = smry.xty_sums[i];
  }
  SET_VECTOR_ELT(out, 0, OCCo);
  SET_VECTOR_ELT(out, 1, ytyo);
  SET_VECTOR_ELT(out, 2, ytxo);
  SET_VECTOR_ELT(out, 3, xtyo);
  UNPROTECT(5);
  return out;
}

extern "C" SEXP Rdevice_mmultiply(SEXP AR, SEXP BR, SEXP a1R, SEXP a2R, SEXP b1R, SEXP b2R){
  int a1 = INTEGER(a1R)[0], a2 = INTEGER(a2R)[0], b1 = INTEGER(b1R)[0], b2 = INTEGER(b2R)[0];
  fvec_d A(REAL(AR), REAL(AR) + a1*a2), B(REAL(BR), REAL(BR) + b1*b2);
  fvec_d big_grid(a2*b2);
  big_matrix_multiply(A, B, big_grid, a1, a2, b1, b2);
  fvec_h big_grid_h(a2*b2);
  thrust::copy(big_grid.begin(), big_grid.end(), big_grid_h.begin());
  SEXP out = PROTECT(allocVector(REALSXP, a2*b2));
  for(int i=0; i<a2*b2; ++i) REAL(out)[i] = big_grid_h[i];
  UNPROTECT(1);
  return out;
}

extern "C" SEXP Rrun_mcmc(SEXP Rdata, SEXP Rpriors, SEXP RmethodPi, SEXP RmethodAlpha, SEXP Rchain, SEXP Rn_iter, SEXP Rn_save_P, SEXP Ridx_save, SEXP Rthin, SEXP Rseed, SEXP Rverbose){
  int verbose = INTEGER(Rverbose)[0];
  std::cout << "verbosity level = " << verbose << std::endl;
  data_t data      = Rdata_wrap(Rdata, verbose-1);
  priors_t priors  = Rpriors_wrap(Rpriors, verbose-1);
  chain_t chain    = Rchain_wrap(Rchain, verbose-1);
  int methodPi     = INTEGER(RmethodPi)[0],
      methodAlpha  = INTEGER(RmethodAlpha)[0],
      n_iter       = INTEGER(Rn_iter)[0],
      thin         = INTEGER(Rthin)[0],
      n_save_P     = INTEGER(Rn_save_P)[0];
  int G_save       = length(Ridx_save), seed = INTEGER(Rseed)[0];
  int n_save_g     = n_iter/thin + (n_iter % thin == 0 ? 0 : 1);
  bool alpha_fixed = methodAlpha == 0;
  
  /* Set thin_P to ensure at least n_save_P draws are saved*/
  int thin_P = n_iter - n_save_P; //in case n_save_P = 1, last iteration is saved
  
  if(n_save_P > 1){
    //if n_save_P is 2 or greater, thin_P is sup(x : x * n_save_P < n_iter)
    thin_P = n_iter/(n_save_P - 1) + (n_iter % (n_save_P - 1) == 0 ? -1 : 0);
  }
  
  samples_t samples(n_save_g, n_save_P, G_save, priors.K, data.V, INTEGER(Ridx_save), alpha_fixed);
  
  std::cout << "Model for pi: ";
  if(methodPi==0){
   std::cout << "Truncated stick-breaking process, ";
  } else if(methodPi==1){
   std::cout << "Symmetric Dirichlet distribution, ";
  }
  if(alpha_fixed){
    std::cout <<"alpha fixed" << std::endl;
  } else{
    std::cout <<"varying alpha" << std::endl;
  }
  
  //instantiate RNGs
  hiprandState *devStates;
  CUDA_CALL(hipMalloc((void **) &devStates, data.G * data.V * sizeof(hiprandState)));
  setup_kernel<<<chain.G, chain.V>>>(seed, chain.G*chain.V, devStates);
  
  
  //progress bar
  boost::progress_display show_progress(n_iter);
  
  for(int i=0; i<n_iter; i++){
    //Gibbs steps
    summary2 summary(chain.K, chain.zeta, data);
    if(verbose > 1){
    std::cout << "Mk:\n";
    printVec(summary.Mk, priors.K, 1);
    std::cout << "occupied:\n";
    printVec(summary.occupied, summary.num_occupied, 1);
    std::cout << "unoccupied:\n";
    printVec(summary.unoccupied, priors.K - summary.num_occupied, 1);
    }
    
    draw_beta(devStates, data, chain, priors, summary, verbose-1);
    if(verbose > 1) {
      std::cout << "beta:\n";
      printVec(chain.beta, data.V, priors.K);
    }
    
    draw_tau2(devStates, chain, priors, data, summary, verbose-1);
    if(verbose > 1){
      std::cout << "tau2:\n";
      printVec(chain.tau2, priors.K, 1);
    }
    if(methodPi == 0){
      draw_pi(devStates, chain, priors, summary, verbose-1);
    } else if(methodPi == 1) {
      draw_pi_SD(devStates, chain, priors, summary, verbose-1);
    }
    if(verbose > 1) {
      std::cout << "pi:\n";
      printVec(chain.pi, priors.K, 1);
    }
    
    if(methodAlpha == 1){
      draw_alpha(chain, priors, verbose-1);
    }
    if(methodAlpha == 2){
      draw_alpha_SD(chain, priors, verbose-1);
    }
    if(!alpha_fixed & verbose > 0) {
      std::cout << "alpha = " << priors.alpha << std::endl;
    }
    
    draw_zeta(devStates, data, chain, priors, verbose-1);
    if(verbose > 1){
      std::cout << "zeta:\n";
      printVec(chain.zeta, data.G, 1);
    }
    
    if(i % thin == 0){
      if(!alpha_fixed){
        samples.save_alpha[samples.step_g] = priors.alpha;
      }
      samples.write_g_samples(chain, summary);
    }
    
    if(i % thin_P == 0 & samples.step_P < n_save_P){
      samples.write_P_samples(chain);
    }
    
    chain.update_means(i+1);
    chain.update_probabilities(i+1);
    ++show_progress;
  }
  
  CUDA_CALL(hipFree(devStates));
  SEXP samples_out = Csamples_wrap(samples, verbose-1);          //PROTECT(7)
  SEXP chain_out   = Cchain_wrap(chain, verbose-1);              //PROTECT(4)
  SEXP out         = PROTECT(allocVector(VECSXP, 2)); //PROTECT(1)
  SET_VECTOR_ELT(out, 0, samples_out);
  SET_VECTOR_ELT(out, 1, chain_out);
  UNPROTECT(12);                                      //7 + 4 + 1
  
  return out;
}
