#include "hip/hip_runtime.h"
#include "../header/gibbs.h"
#include <thrust/scan.h>
#include <thrust/transform_scan.h>

struct log_1m {
  __host__ __device__ double operator()(double &x){
    return log(1-x);
  }
};

struct exp_log_plus {
  __host__ __device__ double operator()(double &x, double &y){
    return exp(log(x) + y);
  }
};

struct modify_gamma_par {
  double N;
  modify_gamma_par(double _N): N(_N){}
  template<typename T>
  __host__ __device__ void operator()(T tup){
    thrust::get<0>(tup) = thrust::get<0>(tup) + 1.0/ 2.0 * thrust::get<1>(tup) * N;
  }
};

void draw_MVNormal(hiprandState *states, fvec_d &beta_hat, fvec_d &chol_prec, fvec_d &beta, priors_t &priors, int verbose = 0){
  //no longer should be passing summary2!
  int K = priors.K;
  int V = priors.V;
  //replace current beta with standard normal draws
  getNormal<<<K, V>>>(states, thrust::raw_pointer_cast(beta.data()));
  
  if(verbose > 1){
    std::cout << "N(0,1) draws:\n";
    printVec(beta, V, K);
  }
  
  //scale occupied betas by t(chol_prec)^-1
  scale_chol_inv(chol_prec, beta, K, V);

  if(verbose > 1){
    std::cout << "scaled draws:\n";
    printVec(beta, V, K);
  }
  
  //shift betas by beta_hat
  thrust::transform(beta_hat.begin(), beta_hat.end(), beta.begin(), beta.begin(), thrust::plus<double>());
  
  if(verbose > 1){
    std::cout << "beta draws:\n";
    printVec(beta, V, K);
  }
}

void draw_tau2(hiprandState *states, chain_t &chain, priors_t &priors, data_t &data, summary2 &smry, int verbose=0){
  fvec_d sse(smry.num_occupied);
  int K = chain.K;
  smry.sumSqErr(sse, chain.beta, data.xtx, --verbose);
  if(verbose > 1){
    std::cout << "sse:\n";
    printVec(sse, smry.num_occupied, 1);
  }
  fvec_d a_d(K, priors.a);
  fvec_d b_d(K, priors.b);
  if(verbose > 1){
    std::cout << "a_d filled:\n";
    printVec(a_d, K, 1);
  }
  // modify gamma parameters for occupied clusters
  typedef thrust::tuple<realIter, intIter> tuple1;
  typedef thrust::zip_iterator<tuple1> zip1;
  tuple1 tup1 = thrust::tuple<realIter, intIter>(a_d.begin(), smry.Mk.begin());
  zip1 zp1 = thrust::zip_iterator<tuple1>(tup1);
  modify_gamma_par f1(data.N);
  thrust::for_each(zp1, zp1 + K, f1);
  
  if(verbose > 1){
    std::cout << "a transformed:\n";
    printVec(a_d, K, 1);
  }
  
  typedef thrust::permutation_iterator<realIter, intIter> FltPermIter;
  FltPermIter b_occ = thrust::permutation_iterator<realIter, intIter>(b_d.begin(), smry.occupied.begin());
  typedef thrust::tuple<FltPermIter, realIter> tuple2;
  typedef thrust::zip_iterator<tuple2> zip2;
  tuple2 tup2 = thrust::tuple<FltPermIter, realIter>(b_occ, sse.begin());
  zip2 zp2 = thrust::zip_iterator<tuple2>(tup2);
  modify_gamma_par f2(1.0);
  if(verbose > 1){
    std::cout << "b filled:\n";
    printVec(b_d, K, 1);
  }
  thrust::for_each(zp2, zp2 + smry.num_occupied, f2);

  if(verbose > 1){
    std::cout << "b transformed:\n";
    printVec(b_d, K, 1);
  }
  // raw pointers
  double *tau2_ptr = thrust::raw_pointer_cast(chain.tau2.data());
  double *a_ptr = thrust::raw_pointer_cast(a_d.data());
  double *b_ptr = thrust::raw_pointer_cast(b_d.data());
  
  //generate
  getGamma<<<K, 1>>>(states, a_ptr, b_ptr, tau2_ptr);
  if(verbose > 1){
    std::cout <<"tau2 immediately after getGamma:\n";
    printVec(chain.tau2, K, 1);
  }
}

void draw_pi(hiprandState *states, chain_t &chain, priors_t &priors, summary2 &summary, int verbose = 0){
  int K = priors.K;
  fvec_d Tk(K);
  fvec_d Mkp1(K);
  fvec_d Vk(K, 1.0);
  if(verbose > 1){
    std::cout << "Tk init:\n";
    printVec(Tk, K, 1);
  }
  thrust::exclusive_scan(summary.Mk.rbegin(), summary.Mk.rend(), Tk.rbegin());
  if(verbose > 1){
    std::cout << "Tk filled:\n";
    printVec(Tk, K, 1);
  }
  thrust::transform(Tk.begin(), Tk.end(), Tk.begin(), thrust::placeholders::_1 + priors.alpha);
  if(verbose > 1){
    std::cout <<"Tk transformed";
    printVec(Tk, K, 1);
  }
  thrust::transform(summary.Mk.begin(), summary.Mk.end(), Mkp1.begin(), thrust::placeholders::_1 + 1.0);
  getBeta<<<K-1, 1>>>(states, thrust::raw_pointer_cast(Mkp1.data()),
                    thrust::raw_pointer_cast(Tk.data()),
                    thrust::raw_pointer_cast(Vk.data()));
  if(verbose > 1){
    std::cout <<"Vk:\n";
    printVec(Vk, K, 1);
  }
  fvec_d Ck(K, 0.0);
  transform_inclusive_scan(Vk.begin(), Vk.end()-1, Ck.begin()+1, log_1m(), thrust::plus<double>());
  if(verbose > 1){
    std::cout << "Ck:\n";
    printVec(Ck, K, 1);
  }
  transform(Vk.begin(), Vk.end(), Ck.begin(), chain.pi.begin(), exp_log_plus());
  if(verbose > 1){
    std::cout << "pi:\n";
    printVec(chain.pi, K, 1);
  }
}

void draw_zeta(hiprandState *states, data_t &data, chain_t &chain, priors_t &priors, int verbose=0){
  fvec_d grid(data.G*priors.K);
  cluster_weights(grid, data, chain);
  if(verbose > 1){
    std::cout << "grid:\n";
    printVec(grid, priors.K, data.G);
  }
  gnl_multinomial(chain.zeta, grid, states, priors.K, data.G);
  if(verbose > 1){
    std::cout << "(inside draw_zeta) zeta:\n";
    printVec(chain.zeta, data.G, 1);
  }
}

void draw_beta(hiprandState *states, data_t &data, chain_t &chain, priors_t &priors, summary2 &smry, int verbose=0){
  fvec_d prec(priors.K * data.V * data.V);
  fvec_d betahat(priors.K * data.V, 0.0);
  
  //get cluster (inv)scales
  construct_prec(prec, data, priors, chain, smry.Mk, --verbose);
  realIter prec_begin = prec.begin();
  realIter prec_end = prec.end();
  chol_multiple(prec_begin, prec_end, data.V, priors.K);
  
  //init betahat with tau2[k] * xty_sum[k] + lambda2 * mu0
  construct_weighted_sum(betahat, smry, priors, chain, --verbose);
  
  beta_hat(prec, betahat, priors.K, data.V);
  draw_MVNormal(states, betahat, prec, chain.beta, priors, --verbose);
}

void draw_pi_SD(hiprandState *states, chain_t &chain, priors_t &priors, summary2 &summary, int verbose = 0){
  int K = priors.K;
  fvec_d a(K);
  fvec_d b(K, 1.0);
  double *a_ptr = thrust::raw_pointer_cast(a.data());
  double *b_ptr = thrust::raw_pointer_cast(b.data());
  double *raw_ptr = thrust::raw_pointer_cast(chain.pi.data());
  getGamma<<<K, 1>>>(states, a_ptr, b_ptr, raw_ptr);
  double sum = thrust::reduce(chain.pi.begin(), chain.pi.end());
  thrust::transform(chain.pi.begin(), chain.pi.end(), chain.pi.begin(), thrust::placeholders::_1 / sum);
}

