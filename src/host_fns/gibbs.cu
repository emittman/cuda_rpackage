#include "../header/gibbs.h"

struct modify_gamma_par{
  template<typename T>
  double operator()(double p, T x){
    return p + x/2;
  }
};

void draw_tau2(hiprandState *states, chain_t &chain, priors_t &prior, data_t &data, summary2 &smry){
  fvec_d sse(smry.num_occupied);
  smry.sumSqErr(sse, chain.beta, data.xtx);
  fvec_d a(prior.K, prior.a);
  fvec_d b(prior.K, prior.b);
  
  // modify gamma parameters for occupied clusters
  typedef thrust::permutation_iterator<intIter, intIter> IntPermIter;
  typedef thrust::permutation_iterator<realIter, intIter> FltPermIter;
  IntPermIter Mk_iter =  thrust::permutation_iterator<intIter, intIter>(smry.Mk.begin(), smry.occupied.begin());
  thrust::transform(a.begin(), a.end(), Mk_iter, a.begin(), modify_gamma_par());
  FltPermIter sse_iter = thrust::permutation_iterator<realIter, intIter>(sse.begin(), smry.occupied.begin());
  thrust::transform(b.begin(), b.end(), sse_iter, b.begin(), modify_gamma_par());
  
  // raw pointers
  double *tau2_ptr = thrust::raw_pointer_cast(chain.tau2.data());
  double *a_ptr = thrust::raw_pointer_cast(a.data());
  double *b_ptr = thrust::raw_pointer_cast(b.data());
  
  // generate
  getGamma<<<chain.K, 1>>>(states, a_ptr, b_ptr, tau2_ptr);
}
