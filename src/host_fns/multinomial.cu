#include "hip/hip_runtime.h"
#include "../header/multinomial.h"
#include "../header/distribution.h"
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

__host__ __device__ double log_sum_exp::operator()(double &x, double &y){
  double M = max(x, y);
  return log(exp(y-M) + exp(x-M)) + M;
}

void normalize_wts(fvec_d &big_grid, int K, int G){
  repEachIter key = getRepEachIter(K, 1);
  log_sum_exp f;
  thrust::inclusive_scan_by_key(key, key + K*G, big_grid.begin(), big_grid.begin(), thrust::equal_to<int>(), f);
}

__host__ __device__ void is_greater::operator()(compare_tup_el Tup){
  if(log(thrust::get<0>(Tup)) > thrust::get<1>(Tup)){
    thrust::get<2>(Tup) = 1;
  } else {
    thrust::get<2>(Tup) = 0;
  }
}

struct exponential{

  __host__ __device__ double operator()(const double &x){
    return exp(x);
  }

};

typedef thrust::permutation_iterator<fvec_d::iterator, repTimesIter> strideIter;


void gnl_multinomial(ivec_d &zeta, fvec_d &probs, hiprandState *states, int K, int G){
  normalize_wts(probs, K, G);
  fvec_d u(G);
  repTimesIter last_row_iter = getRepTimesIter(G, K);
  strideIter strided_iter = thrust::make_permutation_iterator(probs.begin(), last_row_iter);

  thrust::copy(strided_iter, strided_iter + G, u.begin());
  thrust::transform(u.begin(), u.end(), u.begin(), exponential());
  
  std::cout << "this is colSums (?):\n";
  printVec(u, G, 1);
  
  double *u_ptr = thrust::raw_pointer_cast(u.data());
  getUniform<<<G, 1>>>(states, u_ptr);
  ivec_d dummies(K*G);
  gRepEach<realIter>::iterator u_rep = getGRepEachIter(u.begin(), u.end(), K);
  compare_zip zipped = thrust::zip_iterator<compare_tup>(thrust::make_tuple(u_rep, probs.begin(), dummies.begin()));
  is_greater f;
  thrust::for_each(zipped, zipped + K*G, f);
  
  repEachIter colI = getRepEachIter(K, 1);
  thrust::reduce_by_key(colI, colI + K*G, dummies.begin(), thrust::make_discard_iterator(), zeta.begin());
}