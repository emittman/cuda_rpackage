#include "../header/cluster_probability.h"
#include "../header/iter_getter.h"
#include "../header/chain.h"
#include "hipblas.h"

void big_matrix_multiply(fvec_d &A, fvec_d &B, fvec_d &big_grid, int a1, int a2, int b1, int b2){
  double alpha = 1, beta = 0;
  if(a1 != b1) std::cout << "a1 and b1 must be the same (t(A) B = big_grid)\n";
  int lda = a1, ldb = b1, ldc = a2;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasStatus_t status;
  double *A_ptr = thrust::raw_pointer_cast(&(A[0]));
  double *B_ptr = thrust::raw_pointer_cast(&(B[0]));
  double *grid_ptr = thrust::raw_pointer_cast(&(big_grid[0]));
  status = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, a2, b2, a1,
              &alpha, A_ptr, lda,
              B_ptr, ldb,
              &beta, grid_ptr, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cout << "matrix multiply failed\n";
    std::cout << hipGetErrorString(hipGetLastError()));
    std::cout << std::endl;
  }
  hipblasDestroy(handle);
}
