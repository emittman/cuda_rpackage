#include "hip/hip_runtime.h"
#include "../header/distribution.h"

__device__ double rgamma(hiprandState *state, double a, double b){
  //case a >= 1
  double d = a - 1.0 / 3;
  double Y, U, v;
  while(true){
    // Generate a standard normal random variable
    Y = hiprand_normal(state);
    
    v = pow((1 + Y / sqrt(9 * d)), 3);
    
    // Necessary to avoid taking the log of a negative number later
    if(v <= 0) 
      continue;
    
    // Generate a standard uniform random variable
    U = hiprand_uniform(state);
    
    // Accept proposed Gamma random variable under following condition,
    // otherise repeat the loop
    if(log(U) < 0.5 * pow(Y,2) + d * (1 - v + log(v)) ){
      return d * v / b;
    }
  }
}

__device__ double rgamma2(hiprandState *state, double a, double b){
  //case a < 1
  double u, x;
  u = pow(hiprand_uniform(state), 1/a);
  x = rgamma(state, a, b);
  return(u*x);
}

__device__ double rbeta(hiprandState *state,  double a, double b){
  
  double x,y;

  x = rgamma(state, a, 1.0);
  y = rgamma(state, b, 1.0);
  
  return x/(x+y);
}

__global__ void setup_kernel(int seed, hiprandState *states) {
  
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  /* Each thread gets same seed, a different sequence number, no offset */
    
    hiprand_init(seed, id, 0, &states[id]);
}

__global__ void getGamma(hiprandState *states, double *a, double *b, double *result){
  
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  
  if(a[id]>=1){
    result[id] = rgamma(&(states[id]), a[id], b[id]);
  } else {
    result[id] = rgamma2(&(states[id]), a[id], b[id]);
  }
}


__global__ void getBeta(hiprandState *states, double *a, double *b, double *result){
  
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  
  result[id] = rbeta(&(states[id]), a[id], b[id]);
}

__global__ void getUniform(hiprandState *states, double *upper_result){

  int id = threadIdx.x + blockIdx.x * blockDim.x;
  
  upper_result[id] = log(hiprand_uniform(&(states[id]))) + upper_result[id];
}

__global__ void getNormal( hiprandState *states, double *result)
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  
  result[id] = hiprand_normal(&(states[id]));
}
