#include "hip/hip_runtime.h"
#include "../header/cuda_usage.h"

int getDevice(){
	int device;
	hipGetDevice(&device);
  return device;
}

int setDevice(int device) {
	int dev, deviceCount;
	hipGetDeviceCount(&deviceCount);
  if(deviceCount < 1){
    Rprintf("No CUDA-capable GPUs detected.");
    return -1;
  }
  if(device < 0 || device >= deviceCount)
    Rprintf("Warning: invalid device index. Setting device = abs(device) mod deviceCount.\n");
  dev = abs(device) % deviceCount;
	CUDA_CALL(hipSetDevice(dev));
  return dev;
}